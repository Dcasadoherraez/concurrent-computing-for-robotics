/**
*	@Author: https://cuda-tutorial.readthedocs.io/en/latest/tutorials/tutorial01/#tutorial-01-say-hello-to-cuda
*	adjusted by Jahic
*/


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void cuda_hello(){
    printf("Hello World from GPU!\n");
}

int main() 
{
	// TODO: Try changing grid size (e.g., <<1,6>>) to see what happens. 
    cuda_hello<<<2,4>>>(); 
    return 0;
}